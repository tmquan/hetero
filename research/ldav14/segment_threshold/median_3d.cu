#include "hip/hip_runtime.h"
#include "median_3d.hpp"
#include "hip/hip_vector_types.h" 

void median_3d(float* deviceSrc, float* deviceDst, int dimx, int dimy, int dimz, int radius, int halo, hipStream_t stream);

__global__ 
void __median_3d(float* deviceSrc, float* deviceDst, int dimx, int dimy, int dimz, int radius, int halo);

void median_3d(float* deviceSrc, float* deviceDst, int dimx, int dimy, int dimz, int radius, int halo, hipStream_t stream)
{
    dim3 blockDim(8, 8, 8);
    dim3 gridDim(
        (dimx/blockDim.x + ((dimx%blockDim.x)?1:0)),
        (dimy/blockDim.y + ((dimy%blockDim.y)?1:0)),
        (dimz/blockDim.z + ((dimz%blockDim.z)?1:0)) );
    size_t sharedMemSize  = (blockDim.x+2*halo)*(blockDim.y+2*halo)*(blockDim.z+2*halo)*sizeof(float);
    __median_3d<<<gridDim, blockDim, sharedMemSize, stream>>>
     (deviceSrc, deviceDst, dimx, dimy, dimz, radius, halo);
}

inline __device__ __host__ int clamp_mirror(int f, int a, int b)      				
{      																			
    if(f<a) return (a+(a-f));														
    if(f>b) return (b-(f-b));														
    return f;																		
}       																			
#define at(x, y, z, dimx, dimy, dimz) ( clamp_mirror((int)z, 0, dimz-1)*dimy*dimx +       \
                                        clamp_mirror((int)y, 0, dimy-1)*dimx +            \
                                        clamp_mirror((int)x, 0, dimx-1) )                   
__global__ 
void __median_3d(float* deviceSrc, float* deviceDst, int dimx, int dimy, int dimz, int radius, int halo)
{
    extern __shared__ float sharedMemSrc[];                     										
    int  shared_index_1d, global_index_1d, index_1d;                                      										
    int3 shared_index_3d, global_index_3d, index_3d;                                      										
    // Multi batch reading here                                                           										
    int3 sharedMemDim    = make_int3(blockDim.x+2*halo,                                   										
                                     blockDim.y+2*halo,                                  										
                                     blockDim.z+2*halo);                                  										
    int  sharedMemSize   = sharedMemDim.x*sharedMemDim.y*sharedMemDim.z;                  										
    int3 blockSizeDim    = make_int3(blockDim.x+0*halo,                                   										
                                     blockDim.y+0*halo,                                   										
                                     blockDim.z+0*halo);                                  										
    int  blockSize        = blockSizeDim.x*blockSizeDim.y*blockSizeDim.z;                  									
    int  numBatches       = sharedMemSize/blockSize + ((sharedMemSize%blockSize)?1:0);     									
    for(int batch=0; batch<numBatches; batch++)                                           										
    {                                                                                     										
        shared_index_1d  =  threadIdx.z * blockDim.y * blockDim.x +                       										
                            threadIdx.y * blockDim.x +                                    										
                            threadIdx.x +                                                 										
                            blockSize*batch; //Magic is here quantm@unist.ac.kr           										
        shared_index_3d  =  make_int3((shared_index_1d % ((blockDim.y+2*halo)*(blockDim.x+2*halo))) % (blockDim.x+2*halo),		
                                      (shared_index_1d % ((blockDim.y+2*halo)*(blockDim.x+2*halo))) / (blockDim.x+2*halo),		
                                      (shared_index_1d / ((blockDim.y+2*halo)*(blockDim.x+2*halo))) );      					
        global_index_3d  =  make_int3(clamp_mirror(blockIdx.x * blockDim.x + shared_index_3d.x - halo, 0, dimx-1),										
                                      clamp_mirror(blockIdx.y * blockDim.y + shared_index_3d.y - halo, 0, dimy-1), 										
                                      clamp_mirror(blockIdx.z * blockDim.z + shared_index_3d.z - halo, 0, dimz-1) );	
		
        global_index_1d  =  global_index_3d.z * dimy * dimx +                                    								
                            global_index_3d.y * dimx +                                    										
                            global_index_3d.x;                                            										
        if (shared_index_3d.z < (blockDim.z + 2*halo))                                    										
        {                                                                                 										
            if(global_index_3d.z >= 0 && global_index_3d.z < dimz &&                      										
               global_index_3d.y >= 0 && global_index_3d.y < dimy &&                        									
               global_index_3d.x >= 0 && global_index_3d.x < dimx)                        										
            {                                                                             										
                sharedMemSrc[at(shared_index_3d.x, shared_index_3d.y, shared_index_3d.z, sharedMemDim.x, sharedMemDim.y, sharedMemDim.z)] = deviceSrc[global_index_1d];                         
            }                                                                             						
            else                                                                          						
            {                                                                             						
                sharedMemSrc[at(shared_index_3d.x, shared_index_3d.y, shared_index_3d.z, sharedMemDim.x, sharedMemDim.y, sharedMemDim.z)] = -100.0f; 
			}                                                                             
        }                                                                                 
        __syncthreads();                                                                  
    }   

                                                                               
    // Stencil  processing here                                                           
    float result = sharedMemSrc[at(threadIdx.x + halo, threadIdx.y + halo, threadIdx.z + halo, sharedMemDim.x, sharedMemDim.y, sharedMemDim.z)];   
	 // # Viola's method
		// minval = 0
		// maxval = 255
		// pivot  = (minval + maxval)/2.0
		
		// count 	= 0
		// val 	= 0
		// for trial in range(0, 8):
			// radius = 3       
			// cube_iter = make_cube_iter(x, y, z, radius)
			// count 	= 0
			// for point in cube_iter:
				// val = point_query_3d(volume, point)
				// if val > pivot:
					// count = count + 1
		
			// if count < (2*radius+1)*(2*radius+1)*(2*radius+1)/2:
				// maxval = floorf(pivot);      
			// else:
				// minval = floorf(pivot)+1;
			
			// pivot = (minval + maxval)/2.0;
		
		// return floorf(pivot)	
	float minval = 0.0f;                                                                                  
	float maxval = 255.0f;   
	float pivot  = (minval + maxval)/2.0f;
	float val;
	int count  = 0;
	for(int trial=0; trial<8; trial++)
	{
		count = 0;
		for(int z=threadIdx.z+halo-radius; z<=threadIdx.z+halo+radius; z++)
		{
			for(int y=threadIdx.y+halo-radius; y<=threadIdx.y+halo+radius; y++)
			{
				for(int x=threadIdx.x+halo-radius; x<=threadIdx.x+halo+radius; x++)
				{
					val = sharedMemSrc[at(x, y, z, sharedMemDim.x, sharedMemDim.y, sharedMemDim.z)];   
					if(val>pivot)	
						count++;
					
				}
			}
		}
		if(count < (2*radius+1)*(2*radius+1)*(2*radius+1)/2)
			maxval = floorf(pivot);  
		else
			minval = floorf(pivot)+1;
		pivot = (minval + maxval)/2.0f;
	}

	result = floorf(pivot);
	
    // Single pass writing here                                                           
    index_3d       =  make_int3(blockIdx.x * blockDim.x + threadIdx.x,                    
                                blockIdx.y * blockDim.y + threadIdx.y,                    
                                blockIdx.z * blockDim.z + threadIdx.z);                   
    index_1d       =  index_3d.z * dimy * dimx +                                          
                      index_3d.y * dimx +                                                 
                      index_3d.x;                                                         
	                                                                                       
    if (index_3d.z < dimz &&                                                              
        index_3d.y < dimy &&                                                              
        index_3d.x < dimx)                                                                
    {                                                                                     
        deviceDst[index_1d] = result;                                        
    }                                                                                     
}                                                                                         
