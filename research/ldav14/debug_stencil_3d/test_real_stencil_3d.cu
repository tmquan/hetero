#include "hip/hip_runtime.h"
#define BLOCKDIMX 512
#define BLOCKDIMY 1
#define BLOCKDIMZ 1
#define BLOCKDIMXY 		(BLOCKDIMX*BLOCKDIMY)
#define BLOCKDIMXYZ 	(BLOCKDIMX*BLOCKDIMY*BLOCKDIMZ)

#define BLOCKSIZEX 512
#define BLOCKSIZEY 1
#define BLOCKSIZEZ 8

// Use all constants to debug and get the performance
#define DIMX 512
#define DIMY 512
#define DIMZ 512
#define DIMXY (DIMX*DIMY)
#define TOTAL (DIMX*DIMY*DIMZ)


#define NUMTHREADS 		(BLOCKDIMX*BLOCKDIMY*BLOCKDIMZ)
#define HALO 			1
#define OPENEDDIMX  	(BLOCKSIZEX+2*HALO)
#define OPENEDDIMY  	(BLOCKSIZEY+2*HALO)
#define OPENEDDIMZ  	(BLOCKSIZEZ+2*HALO)
#define OPENEDDIMXY 	(OPENEDDIMX*OPENEDDIMY)
#define OPENEDDIMXYZ  	(OPENEDDIMX*OPENEDDIMY*OPENEDDIMZ)
#define CLOSEDDIMX  	(BLOCKSIZEX)
#define CLOSEDDIMY  	(BLOCKSIZEY)
#define CLOSEDDIMZ  	(BLOCKSIZEZ)
#define CLOSEDDIMXY 	(CLOSEDDIMX*CLOSEDDIMY)
#define CLOSEDDIMXYZ  	(CLOSEDDIMX*CLOSEDDIMY*CLOSEDDIMZ)
#define NUMREADING  	((OPENEDDIMXYZ / NUMTHREADS) + ((OPENEDDIMXYZ%NUMTHREADS)?1:0))
#define NUMWRITING  	((CLOSEDDIMXYZ / NUMTHREADS) + ((CLOSEDDIMXYZ%NUMTHREADS)?1:0))

// #define CORRECTNESS_DATA
#define CORRECTNESS_HEAT
// #define myclamp(x, value, tx, fx) {return ((x)==(value)) ? (tx):(fx)}
#define C0 0.25f
#define C1 0.50f


#include <iostream>
#include <fstream>
#include <sstream>
#include <iomanip>      // std::setfill, std::setw
#include <string>
// #include <sys/ioctl.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
// #include <gpu_timer.hpp>

using namespace std;
////////////////////////////////////////////////////////////////////////////////////////////////////
#define checkLastError() {                                          				\
	hipError_t error = hipGetLastError();                               			\
	int id; 																		\
	hipGetDevice(&id);																\
	if(error != hipSuccess) {                                         				\
		printf("Cuda failure error in file '%s' in line %i: '%s' at device %d \n",	\
			__FILE__,__LINE__, hipGetErrorString(error), id);			      	 	\
		exit(EXIT_FAILURE);  														\
	}                                                               				\
}
////////////////////////////////////////////////////////////////////////////////////////////////////

#define checkReadFile(filename, pData, size) {                    					\
		fstream *fs = new fstream;													\
		fs->open(filename, ios::in|ios::binary);									\
		if (!fs->is_open())															\
		{																			\
			printf("Cannot open file '%s' in file '%s' at line %i\n",				\
			filename, __FILE__, __LINE__);											\
			return 1;																\
		}																			\
		fs->read(reinterpret_cast<char*>(pData), size);								\
		fs->close();																\
		delete fs;																	\
	}																			

////////////////////////////////////////////////////////////////////////////////////////////////////
#define checkWriteFile(filename, pData, size) {                    					\
		fstream *fs = new fstream;													\
		fs->open(filename, ios::out|ios::binary);									\
		if (!fs->is_open())															\
		{																			\
			fprintf(stderr, "Cannot open file '%s' in file '%s' at line %i\n",		\
			filename, __FILE__, __LINE__);											\
			return 1;																\
		}																			\
		fs->write(reinterpret_cast<char*>(pData), size);							\
		fs->close();																\
		delete fs;																	\
	}
////////////////////////////////////////////////////////////////////////////////////////////////////


// #define at(x, y, z, DIMX, DIMY, DIMZ) ( clamp((int)(z), 0, DIMZ-1)*DIMY*DIMX +  	\
                                        // clamp((int)(y), 0, DIMY-1)*DIMX +       	\
                                        // clamp((int)(x), 0, DIMX-1) )                   
#define at(x, y, z) ( (z)*DIMXY + (y)*DIMX  +  (x) )                   

										
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void heatflow_global(float *src, float *dst)
{
	int  closed_index_1d, offset_index_1d, global_index_1d;
	int3 closed_index_3d, offset_index_3d, global_index_3d;
	offset_index_3d  = make_int3(blockIdx.x * BLOCKSIZEX, 
								 blockIdx.y * BLOCKSIZEY,
								 blockIdx.z * BLOCKSIZEZ);
	float nextZ, currZ, prevZ;			 
	float nextY, currY, prevY;			 
	float nextX, currX, prevX;			 
	#pragma unroll
	for(int thisWriting=0; thisWriting<NUMWRITING; thisWriting++)
	{
		closed_index_1d = threadIdx.z * BLOCKDIMXY +
						  threadIdx.y * BLOCKDIMX +
						  threadIdx.x + 
		// closed_index_1d =  threadIdx.x + 
						   thisWriting*NUMTHREADS;
		closed_index_3d = make_int3((closed_index_1d % CLOSEDDIMXY % CLOSEDDIMX),		
								    (closed_index_1d % CLOSEDDIMXY / CLOSEDDIMX),		
									(closed_index_1d / CLOSEDDIMXY) );  
		global_index_3d = make_int3((offset_index_3d.x + closed_index_3d.x),
									(offset_index_3d.y + closed_index_3d.y),
									(offset_index_3d.z + closed_index_3d.z) );
		
	
		
		if(global_index_3d.z > 0 && global_index_3d.z < (DIMZ-1) &&	
		   global_index_3d.y > 0 && global_index_3d.y < (DIMY-1) &&
		   global_index_3d.x > 0 && global_index_3d.x < (DIMX-1) ) 
		{
			global_index_1d = global_index_3d.z * DIMXY +
							  global_index_3d.y * DIMX +
							  global_index_3d.x;
						  
			// dst[at(global_index_3d.x, global_index_3d.y, global_index_3d.z, DIMX, DIMY, DIMZ)] 
			// dst[global_index_1d] 
			// = C0 * (src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z+0)])+
			  // C1 * (src[at(global_index_3d.x-1, global_index_3d.y+0, global_index_3d.z+0)] +
					// src[at(global_index_3d.x+1, global_index_3d.y+0, global_index_3d.z+0)] +
					// src[at(global_index_3d.x+0, global_index_3d.y-1, global_index_3d.z+0)] +
					// src[at(global_index_3d.x+0, global_index_3d.y+1, global_index_3d.z+0)] +
					// src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z-1)] +
					// src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z+1)]);
			// if(thisWriting==0)
			// {
			
				nextZ = src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z+1)];
				prevZ = (thisWriting == 0) ? src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z-1)] : currZ;
				currZ = (thisWriting == 0) ? src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z+0)] : nextZ;
				
				prevX = src[at(global_index_3d.x-1, global_index_3d.y+0, global_index_3d.z+0)];
				nextX = src[at(global_index_3d.x+1, global_index_3d.y+0, global_index_3d.z+0)];
				
				
				prevY = src[at(global_index_3d.x+0, global_index_3d.y-1, global_index_3d.z+0)];
				nextY = src[at(global_index_3d.x+0, global_index_3d.y+1, global_index_3d.z+0)];
				
				dst[global_index_1d] 
				= C0 * (currZ)+ C1 * (prevX + nextX + prevY + nextY + prevZ + nextZ);
				
			// }
			// else
			// {
				// prevZ = currZ;
				// currZ = nextZ;
				
				// // prevZ = src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z-1)];
				// // nextZ = src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z+1)];
				
				// prevX = src[at(global_index_3d.x-1, global_index_3d.y+0, global_index_3d.z+0)];
				// nextX = src[at(global_index_3d.x+1, global_index_3d.y+0, global_index_3d.z+0)];
				
				// // prevY = currY;
				// // currY = nextY;
				// // currY = src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z+0)];
				// // prevY = src[at(global_index_3d.x+0, global_index_3d.y-1, global_index_3d.z+0)];
				// // nextY = src[at(global_index_3d.x+0, global_index_3d.y+1, global_index_3d.z+0)];
				
				// dst[global_index_1d] 
				// = C0 * (currZ)+
				  // C1 * (prevX +
						// nextX +
						// prevY +
						// nextY +
						// prevZ +
						// nextZ);
			// }
			// // prevZ = currZ;
			// // currZ = nextZ;
			// // __threadfence_block();
		}
	}
}


////////////////////////////////////////////////////////////////////////////////////////////////////
// __global__
// void heatflow_shared(float *src, float *dst)
// {
	// int  opened_index_1d, closed_index_1d, offset_index_1d, global_index_1d;
	// int3 opened_index_3d, closed_index_3d, offset_index_3d, global_index_3d;
	// offset_index_3d  = make_int3(blockIdx.x * BLOCKSIZEX, 
								 // blockIdx.y * BLOCKSIZEY,
								 // blockIdx.z * BLOCKSIZEZ);
								 
	// __shared__ float sharedMem[OPENEDDIMZ][OPENEDDIMY][OPENEDDIMX];
	// float result;
	
	// int index = threadIdx.z * blockDim.y * blockDim.x +
				// threadIdx.y * blockDim.x +                                   										
				// threadIdx.x;
	// #pragma unroll
	// for(int thisReading=0; thisReading<NUMREADING; thisReading++)
	// {
		// // opened_index_1d = threadIdx.z * blockDim.y * blockDim.x +
						  // // threadIdx.y * blockDim.x +
						  // // threadIdx.x + 
		// opened_index_1d = index +
						  // thisReading * NUMTHREADS;
		// opened_index_3d = make_int3((opened_index_1d % OPENEDDIMXY % OPENEDDIMX),		
								    // (opened_index_1d % OPENEDDIMXY / OPENEDDIMX),		
									// (opened_index_1d / OPENEDDIMXY) );  
		// global_index_3d = make_int3((offset_index_3d.x + opened_index_3d.x - HALO),
									// (offset_index_3d.y + opened_index_3d.y - HALO),
									// (offset_index_3d.z + opened_index_3d.z - HALO) );
		// global_index_1d = global_index_3d.z * DIMY * DIMX +
						  // global_index_3d.y * DIMX +
						  // global_index_3d.x;
		// if(opened_index_3d.z < OPENEDDIMZ)
		// {
			// if(global_index_3d.z >= 0 && global_index_3d.z < (DIMZ) &&	
			   // global_index_3d.y >= 0 && global_index_3d.y < (DIMY) &&
		       // global_index_3d.x >= 0 && global_index_3d.x < (DIMX) ) 
			// {
				// sharedMem[opened_index_3d.z][opened_index_3d.y][opened_index_3d.x]
				// = src[global_index_1d];
			// }
		// }
		
	// }
	// __syncthreads();
	
	// #pragma unroll
	// for(int thisWriting=0; thisWriting<NUMWRITING; thisWriting++)
	// {
		// // closed_index_1d = threadIdx.z * blockDim.y * blockDim.x +
						  // // threadIdx.y * blockDim.x +
						  // // threadIdx.x + 
		// closed_index_1d = index +
						  // thisWriting * NUMTHREADS;
		// closed_index_3d = make_int3((closed_index_1d % CLOSEDDIMXY % CLOSEDDIMX),		
								    // (closed_index_1d % CLOSEDDIMXY / CLOSEDDIMX),		
									// (closed_index_1d / CLOSEDDIMXY) );  
		// global_index_3d = make_int3((offset_index_3d.x + closed_index_3d.x),
									// (offset_index_3d.y + closed_index_3d.y),
									// (offset_index_3d.z + closed_index_3d.z) );
		// global_index_1d = global_index_3d.z * DIMY * DIMX +
						  // global_index_3d.y * DIMX +
						  // global_index_3d.x;
		
		// result = C0 * (sharedMem[closed_index_3d.z+HALO+0][closed_index_3d.y+HALO+0][closed_index_3d.x+HALO+0])+
				 // C1 * (sharedMem[closed_index_3d.z+HALO+0][closed_index_3d.y+HALO+0][closed_index_3d.x+HALO-1] +
					   // sharedMem[closed_index_3d.z+HALO+0][closed_index_3d.y+HALO+0][closed_index_3d.x+HALO+1] +
					   // sharedMem[closed_index_3d.z+HALO+0][closed_index_3d.y+HALO-1][closed_index_3d.x+HALO+0] +
					   // sharedMem[closed_index_3d.z+HALO+0][closed_index_3d.y+HALO+1][closed_index_3d.x+HALO+0] +
					   // sharedMem[closed_index_3d.z+HALO-1][closed_index_3d.y+HALO+0][closed_index_3d.x+HALO+0] +
					   // sharedMem[closed_index_3d.z+HALO+1][closed_index_3d.y+HALO+0][closed_index_3d.x+HALO+0]);
		// if(global_index_3d.z > 0 && global_index_3d.z < (DIMZ-1) &&	
		   // global_index_3d.y > 0 && global_index_3d.y < (DIMY-1) &&
		   // global_index_3d.x > 0 && global_index_3d.x < (DIMX-1) ) 
		// {
			// dst[global_index_1d] 
			// = result;
			// // = C0 * (src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z+0, DIMX, DIMY, DIMZ)])+
			  // // C1 * (src[at(global_index_3d.x-1, global_index_3d.y+0, global_index_3d.z+0, DIMX, DIMY, DIMZ)] +
					// // src[at(global_index_3d.x+1, global_index_3d.y+0, global_index_3d.z+0, DIMX, DIMY, DIMZ)] +
					// // src[at(global_index_3d.x+0, global_index_3d.y-1, global_index_3d.z+0, DIMX, DIMY, DIMZ)] +
					// // src[at(global_index_3d.x+0, global_index_3d.y+1, global_index_3d.z+0, DIMX, DIMY, DIMZ)] +
					// // src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z-1, DIMX, DIMY, DIMZ)] +
					// // src[at(global_index_3d.x+0, global_index_3d.y+0, global_index_3d.z+1, DIMX, DIMY, DIMZ)]);
		// }
	// }
// }

////////////////////////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
	printf("-----------------------------------------------------------------------\n");
	srand(time(NULL)); // for random number generator

	hipSetDevice(0);checkLastError();
	hipDeviceReset();checkLastError();
	// Specify dimensions

	// Allocate host memory
	float *h_src = new float[TOTAL];
	float *h_dst = new float[TOTAL];
	
	// Allocate device memory
	float *d_src;
	float *d_dst;

	hipMalloc((void**)&d_src, TOTAL*sizeof(float));		checkLastError();
	hipMalloc((void**)&d_dst, TOTAL*sizeof(float));		checkLastError();
	
	// Initialize the image source
	for(int z=0; z<DIMZ; z++)
	{
		for(int y=0; y<DIMY; y++)
		{
			for(int x=0; x<DIMX; x++)
			{
				h_src[z*DIMY*DIMX+y*DIMX+x] = (float)( (int)rand() % 10); // 7;
			}
		}
	}
	
	// Transferring to the device memory
	hipMemcpy(d_src, h_src, TOTAL*sizeof(float), hipMemcpyHostToDevice); checkLastError();
	hipMemset(d_dst, 0, TOTAL*sizeof(float));checkLastError();
	
	// parameters for performance eval
	double flops, gbps, nops, nbp;
	nbp = 8*4; // # of bytes transferred per point
	nops = 8.; // # of flops per point
	int iter = 20;
	int rightData = 1;
	int rightHeat = 1;
	/// Verify the correctness of data
// #ifdef CORRECTNESS_DATA
	hipMemcpy(d_dst, d_src, TOTAL*sizeof(float), hipMemcpyDeviceToDevice); checkLastError();
	hipMemcpy(h_dst, d_dst, TOTAL*sizeof(float), hipMemcpyDeviceToHost); checkLastError();
	for(int z=0; z<DIMZ && rightData; z++)
	{
		for(int y=0; y<DIMY && rightData; y++)
		{
			for(int x=0; x<DIMX && rightData; x++)
			{
				if(h_src[z*DIMY*DIMX+y*DIMX+x] != h_dst[z*DIMY*DIMX+y*DIMX+x])
				{
					printf("Data does not match at x: %d, y: %d, z: %d\n", x, y, z);
					rightData = 0;
					// goto cleanup_data;
				}
			}
		}
	}
	if(rightData)		printf("Data is correct.\n");
// cleanup_data:
// #endif
	// grid construction
	dim3 numThreads(BLOCKDIMX, BLOCKDIMY, BLOCKDIMZ); //Dim
	dim3 numBlocks((DIMX/BLOCKSIZEX)+((DIMX%BLOCKSIZEX)?1:0),	//Size  for ILP
				   (DIMY/BLOCKSIZEY)+((DIMY%BLOCKSIZEY)?1:0),
				   (DIMZ/BLOCKSIZEZ)+((DIMZ%BLOCKSIZEZ)?1:0));
	hipMemset(d_dst, 0, TOTAL*sizeof(float));checkLastError(); // Reset the result
	memset(h_dst, 0, TOTAL*sizeof(float));
	printf("Blockdim (%03d, %03d, %03d); Blocksize (%03d, %03d, %03d);\n",
		BLOCKDIMX, BLOCKDIMY, BLOCKDIMZ, BLOCKSIZEX, BLOCKSIZEY, BLOCKSIZEZ);
	// launch kernel
	// GpuTimer gpu_timer;
	// gpu_timer.Start();
	hipEvent_t begin, end;

	hipEventCreate(&begin);
	hipEventCreate(&end);
	hipEventRecord(begin, 0);
	for(int n=0; n<iter; n++)
	{
		heatflow_global<<<numBlocks, numThreads>>>(d_src, d_dst);
		// heatflow_shared<<<numBlocks, numThreads>>>(d_src, d_dst);
	}
	// gpu_timer.Stop();
	hipDeviceSynchronize();
	hipEventRecord(end, 0);
	hipEventSynchronize(end);

	float msec;
	hipEventElapsedTime(&msec, begin, end);
	checkLastError();
	
	// float msec = gpu_timer.Elapsed();
	gbps = nbp*DIMX*DIMY*DIMZ/(msec/1000.)/(1024.*1024.*1024.)*(double)iter;
	flops = nops*DIMX*DIMY*DIMZ/(msec/1000.)/(1024.*1024.*1024.)*(double)iter;
	printf("Computing time : %.3f msec, Device memory bandwidth : %.3f GB/s, GFLOPS : %.3f\n", 		msec, gbps, flops);

	float* h_ref = new float[DIMX*DIMY*DIMZ];
	float tmp, result;
// #ifdef CORRECTNESS_HEAT
	/// Verify the correctness of heat flow, no check at boundary
	// Golden result

	for(int z=1; z<(DIMZ-1); z++)
	{
		for(int y=1; y<(DIMY-1); y++)
		{
			for(int x=1; x<(DIMX-1); x++)
			{
				result = C0 * (h_src[at(x+0, y+0, z+0)])+
						 C1 * (h_src[at(x-1, y+0, z+0)] +
							   h_src[at(x+1, y+0, z+0)] +
							   h_src[at(x+0, y-1, z+0)] +
							   h_src[at(x+0, y+1, z+0)] +
							   h_src[at(x+0, y+0, z-1)] +
							   h_src[at(x+0, y+0, z+1)]);		
				h_ref[at(x+0, y+0, z+0)] 	= result;
			}
		}
	} 

	// Transferring to the host memory
	hipMemcpy(h_dst, d_dst, TOTAL*sizeof(float), hipMemcpyDeviceToHost); checkLastError();
	// Compare result

	for(int z=1; z<(DIMZ-1) && rightHeat; z++)
	{
		for(int y=1; y<(DIMY-1) && rightHeat; y++)
		{
			for(int x=1; x<(DIMX-1) && rightHeat; x++)
			{
				if(h_ref[z*DIMY*DIMX+y*DIMX+x] != h_dst[z*DIMY*DIMX+y*DIMX+x])
				{
					printf("Solution does not match at x: %d, y: %d, z: %d\n", x, y, z);
					printf("h_ref (%04.4f), h_dst (%04.4f)\n", 
						h_ref[z*DIMXY+y*DIMX+x], 
						h_dst[z*DIMXY+y*DIMX+x]);
					rightHeat = 0;
					// goto cleanup_heat;
				}
			}
		}
	}
	if(rightHeat)	printf("Solution is correct.\n");
// cleanup_heat:
// #endif
	///!!! Print line
	// struct winsize w;
    // ioctl(0, TIOCGWINSZ, &w);
	// for(int k=0; k<w.ws_col; k++) 
		// printf("-");
	printf("\n");
	checkLastError();	
// cleanup:
	hipFree(d_src);
	hipFree(d_dst);
	free(h_src);
	free(h_dst);
	free(h_ref);
	return 0;
}