//Written by professor Won-Ki Jeong
// wkjeong@unist.ac.kr
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>
//#include <helper_math.h>
//#include <cutil_inline.h>

#define DIMX 512
#define DIMY 512
#define DIMZ 512
#define SLICE DIMX*DIMY
#define C0 0.25
#define C1 0.5

#define ILP 8


int clamp(int a, int b, int c)
{
   int ret = a;
   if(a < b) ret = b;
   if(a > c) ret = c;
   return ret;
}

#define at(x, y, z, DIMX, DIMY, DIMZ) ( clamp((int)(z), 0, DIMZ-1)*DIMY*DIMX +       \
                                        clamp((int)(y), 0, DIMY-1)*DIMX +            \
                                        clamp((int)(x), 0, DIMX-1) )                   

										
__global__ void test(float *a, float *b, float *c)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;

	// only first warp works 10x more but the total running time is same because
	// block running time is bound by max warp running time
	if(threadIdx.x <32)
	{
		for(int i=0; i<10; i++)
		{
			int idx = DIMX*(DIMY*z + y) + x;
			c[idx] += a[idx]*b[idx]*0.1;
		}
	}

}

__global__ void vectorMul(float *a, float *b, float *c)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;

#pragma unroll 
	int offset = DIMX*y + x;
	for(int i=0; i<ILP; i++)
	{
		int idx = DIMX*DIMY*(ILP*z+i) + offset;

		float _a = a[idx];
		float _b = b[idx];

		float _c = _a*_b;
		
		c[idx] = _c;
	}
}

/*
__global__ void init()
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;

	for(int i=0; i<ILP; i++)
	{
		v[x][y][z+i] = 1.0;
	}
}
*/

//int nboffset[] = { 0, 1, -1 };

__global__ void heatflow(float *a, float *c)
{
	int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;
	int z = blockIdx.z*blockDim.z + threadIdx.z;


	int offset = DIMX*y + x;
	//int idx = SLICE*z + offset;

	int idx, t, nz;
	float center, left, right, top, bottom, front, back;

#pragma unroll 
	for(int i=0; i<ILP; i++)
	{
		//idx += SLICE*i;

		nz = ILP*z+i;
		idx = SLICE*(nz) + offset;
		
		center = a[idx];

		// int t = (x == 0) ? 0 : -1;
		t = (x == 0) ? 0 : -1;
		left = a[idx + t];
	
		t = (x == (DIMX-1)) ? 0 : 1;
		right = a[idx + t];

		t = (y == 0) ? 0 : -DIMX;
		top = a[idx + t];

		t = (y == (DIMY-1)) ? 0 : DIMX;
		bottom = a[idx + t];

		t = (nz == 0) ? 0 : -SLICE;
		front = a[idx + t];

		t = (nz == (DIMZ-1)) ? 0 : SLICE;
		back = a[idx + t];
		
		c[idx] = C0*center + C1*(left + right + top + bottom + front + back);
	}
}



int main()
{
	//srand ( time(NULL) );

	//int gpuid = rand() % 8;
	//printf("Assigned GPU ID: %d\n", gpuid);
	//cudaSetDevice( gpuid ); 

	// Allocate GPU memory	
	float *d_a, *d_b, *d_c, *h_c;


	h_c = (float*)malloc(sizeof(float)*DIMX*DIMY*DIMZ);
	hipMalloc((void**)&(d_a),sizeof(float)*DIMX*DIMY*DIMZ);
	hipMalloc((void**)&(d_b),sizeof(float)*DIMX*DIMY*DIMZ);
	hipMalloc((void**)&(d_c),sizeof(float)*DIMX*DIMY*DIMZ);
	
	for(int i=0; i<DIMX*DIMY*DIMZ; i++) h_c[i] = (float)( (int)rand() % 10); // 7;

	hipMemcpy(d_a, h_c, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice);
	//cudaMemcpy(d_b, h_c, sizeof(float)*DIMX*DIMY*DIMZ, cudaMemcpyHostToDevice);
	hipMemcpy(d_c, h_c, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyHostToDevice);
	

	// copy host to device
	//cudaMemcpy(d_tree.info, h_tree.info, sizeof(stat), cudaMemcpyHostToDevice);

	hipEvent_t begin, end;

	hipEventCreate(&begin);
	hipEventCreate(&end);
	hipEventRecord(begin, 0);


	// call your kernel here
	dim3 dimB = dim3(512,1,8/ILP); // block size
	dim3 dimG = dim3(DIMX/dimB.x,DIMY/dimB.y,DIMZ/(dimB.z*ILP));

	// parameters for performance eval
	double flops, gbps, nops, nbp;
	int iter = 20;

#define HEATFLOW //VECMUL//

#ifdef VECMUL
	nbp = 4*3; // # of bytes transferred per point
	nops = 1.; // # of flops per point
	
	for(int i=0; i<iter; i++)
	{
		vectorMul<<<dimG,dimB>>>(d_a, d_b, d_c);
	}
#endif

#ifdef HEATFLOW
	nbp = 8*4; // # of bytes transferred per point
	nops = 8.; // # of flops per point
	
	// initialize global memory
	//init<<<dimG,dimB>>>();

	for(int i=0; i<iter; i++)
	{
		heatflow<<<dimG,dimB>>>(d_a, d_c);
//		heatflow_shared<<<dimG,dimB>>>(d_a, d_c);
	}


#endif


	//
	hipDeviceSynchronize();

	hipEventRecord(end, 0);
	hipEventSynchronize(end);

	float totalTime;
	hipEventElapsedTime(&totalTime, begin, end);

	gbps = nbp*DIMX*DIMY*DIMZ/(totalTime/1000.)/(1024.*1024.*1024.)*(double)iter;
	flops = nops*DIMX*DIMY*DIMZ/(totalTime/1000.)/(1024.*1024.*1024.)*(double)iter;
	printf("Computing time : %.3f msec, Device memory bandwidth : %.3f GB/s, GFLOPS : %.3f\n", totalTime, gbps, flops);

	/// Verify the correctness
	float* h_r = new float[DIMX*DIMY*DIMZ];
	float tmp, result;
	for(int z=0; z<DIMZ; z++)
	{
		for(int y=0; y<DIMY; y++)
		{
			for(int x=0; x<DIMX; x++)
			{
				tmp = C1 *    ( h_c[at(x + 1, y + 0, z + 0, DIMX, DIMY, DIMZ)] +
								h_c[at(x - 1, y + 0, z + 0, DIMX, DIMY, DIMZ)] +
								h_c[at(x + 0, y + 1, z + 0, DIMX, DIMY, DIMZ)] +
								h_c[at(x + 0, y - 1, z + 0, DIMX, DIMY, DIMZ)] +
								h_c[at(x + 0, y + 0, z + 1, DIMX, DIMY, DIMZ)] +
								h_c[at(x + 0, y + 0, z - 1, DIMX, DIMY, DIMZ)] );
				result = C0*h_c[at(x + 0, y + 0, z + 0, DIMX, DIMY, DIMZ)]  + tmp;			
				h_r[at(x, y, z, DIMX, DIMY, DIMZ)] 	= result;
			}
		}
	}
	hipMemcpy(h_c, d_c, sizeof(float)*DIMX*DIMY*DIMZ, hipMemcpyDeviceToHost);

	for(int x=0; x<DIMX; x++)
	{
		for(int y=0; y<DIMY; y++)
		{
			for(int z=0; z<DIMZ; z++)
			{		
				if(h_r[z*DIMY*DIMX+y*DIMX+x] != h_c[z*DIMY*DIMX+y*DIMX+x])
				{
					printf("Solution does not match at x: %d, y: %d, z: %d\n", x, y, z);
					printf("h_r (%04.4f), h_c (%04.4f)\n", 
						h_r[z*DIMY*DIMX+y*DIMX+x], 
						h_c[z*DIMY*DIMX+y*DIMX+x]);
					return -1;
					// goto cleanup;
				}
			}
		}
	}
	printf("Solution is correct.\n");
	
	printf("Value %f\n", h_c[0]);
	
	hipFree( d_a );
	hipFree( d_b );
	hipFree( d_c );
	

	return 0;
}
