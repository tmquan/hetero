#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
////////////////////////////////////////////////////////////////////////////////////////////////////
#include "add.hpp"
#include "timer.hpp"
#include "utility.hpp"
#include "hip/hip_vector_types.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
using namespace std;
using namespace csmri;
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__ void	warmUp()	{}
////////////////////////////////////////////////////////////////////////////////////////////////////	
int main(int argc, char** argv)
{
	hipSetDevice(0);
	hipDeviceReset();
	warmUp<<<1, 1>>>();

	int dimx = 128;
	int dimy = 128;
	int dimz = 256;
	/// Calculate the total size
	int total = dimx*dimy*dimz;
	
	////////////////////////////////////////////////////////////////////////////////////////////////////
	GpuTimer timer;
	float2 *h_A, *h_B, *h_C, *h_D;
	float2 *d_A, *d_B, *d_C;
	
	fstream hAddFile, dAddFile;
	h_A = new float2[total];
	h_B = new float2[total];
	h_C = new float2[total];
	h_D = new float2[total];
	//Generate data for testing
	srand(time(NULL));
	for(int i=0; i<total; i++)
	{
		h_A[i] = make_float2( (float)rand()/RAND_MAX, (float)rand()/RAND_MAX );
		h_B[i] = make_float2( (float)rand()/RAND_MAX, (float)rand()/RAND_MAX );
	}
	////////////////////////////////////////////////////////////////////////////////////////////////////
	//CPU addition
	for(int i=0; i<total; i++)
	{
		h_C[i] = h_A[i] + h_B[i];
	}

	checkWriteFile("hAdd.bin", h_C, total*sizeof(float2));
	////////////////////////////////////////////////////////////////////////////////////////////////////
	//GPU addition
	hipMalloc((void**)&d_A, total*sizeof(float2));
	hipMalloc((void**)&d_B, total*sizeof(float2));
	hipMalloc((void**)&d_C, total*sizeof(float2));
	hipMemcpy(d_A, h_A, total*sizeof(float2), hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, total*sizeof(float2), hipMemcpyHostToDevice);
	
	timer.Start();
	add(d_A, d_B, d_C, dimx, dimy, dimz);
	timer.Stop();
	
	printf("Addition: %4.4f ms\n", timer.Elapsed());
	
	hipMemcpy(h_D, d_C, total*sizeof(float2), hipMemcpyDeviceToHost);

	checkWriteFile("dAdd.bin", h_D, total*sizeof(float2));
	////////////////////////////////////////////////////////////////////////////////////////////////////
	return 0;
}