#include "hip/hip_runtime.h"
#include "add.hpp"
#include <hip/hip_complex.h>
#include "utility.hpp"
#include "hip/hip_vector_types.h"
////////////////////////////////////////////////////////////////////////////////////////////////////
namespace csmri
{
////////////////////////////////////////////////////////////////////////////////////////////////////
#ifndef blockDimx
#define blockDimx 16		
#endif

#ifndef blockDimy
#define blockDimy 16		
#endif

#ifndef blockDimz
#define blockDimz 1		
#endif
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void __add(
	float2* inA,	
	float2* inB,	
	float2* out,	
	int dimx,
	int dimy,
	int dimz)
{
	//3D global index
	int3 idx = make_int3(
		blockIdx.x*blockDim.x+threadIdx.x,
		blockIdx.y*blockDim.y+threadIdx.y,
		blockIdx.z*blockDim.z+threadIdx.z);
	
	//1D global index
	int index 	= 	idx.z*dimy*dimx		
				+	idx.y*dimx				
				+	idx.x;				
									  
	//Check valid indices
	if (idx.x >= dimx || idx.y >= dimy || idx.z >= dimz)
		return;
	
	//Do computing
	out[index] = hipCaddf(inA[index], inB[index]);
}
////////////////////////////////////////////////////////////////////////////////////////////////////
void add(
	float2* inA,	
	float2* inB,	
	float2* out,	
	int dimx,
	int dimy,
	int dimz)
{
	dim3 numBlocks(
		(dimx/blockDimx + ((dimx%blockDimx)?1:0)),
		(dimy/blockDimy + ((dimy%blockDimy)?1:0)),
		(dimz/blockDimz + ((dimz%blockDimz)?1:0)) );
	dim3 numThreads(blockDimx, blockDimy, blockDimz);
	__add<<<numBlocks, numThreads>>>(inA, inB, out, dimx, dimy, dimz);
}
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void __add(
	float2* inA,	
	float2* inB,	
	float2* inC,
	float2* inD,
	float2* inE,
	float2* out,	
	int dimx,
	int dimy,
	int dimz)
{
	//3D global index
	int3 idx = make_int3(
		blockIdx.x*blockDim.x+threadIdx.x,
		blockIdx.y*blockDim.y+threadIdx.y,
		blockIdx.z*blockDim.z+threadIdx.z);
	
	//1D global index
	int index 	= 	idx.z*dimy*dimx		
				+	idx.y*dimx				
				+	idx.x;				
									  
	//Check valid indices
	if (idx.x >= dimx || idx.y >= dimy || idx.z >= dimz)
		return;
	
	//Do computing
	out[index] = inA[index] + inB[index] + inC[index] + inD[index] + inE[index];
}
////////////////////////////////////////////////////////////////////////////////////////////////////
void add(
	float2* inA,	
	float2* inB,	
	float2* inC,
	float2* inD,
	float2* inE,
	float2* out,	
	int dimx,
	int dimy,
	int dimz,
	hipStream_t stream)
{
	dim3 numBlocks(
		(dimx/blockDimx + ((dimx%blockDimx)?1:0)),
		(dimy/blockDimy + ((dimy%blockDimy)?1:0)),
		(dimz/blockDimz + ((dimz%blockDimz)?1:0)) );
	dim3 numThreads(blockDimx, blockDimy, blockDimz);
	// __add<<<numBlocks, numThreads>>>(inA, inB, inC, inD, inE, out, dimx, dimy, dimz);
	__add<<<numBlocks, numThreads, 0, stream>>>(inA, inB, inC, inD, inE, out, dimx, dimy, dimz);
}
////////////////////////////////////////////////////////////////////////////////////////////////////
}