#include "hip/hip_runtime.h"
#include "ddt.hpp"
#include "utility.hpp"
#include "hip/hip_vector_types.h"
namespace csmri
{
////////////////////////////////////////////////////////////////////////////////////////////////////
#ifndef blockDimx
#define blockDimx 16		
#endif

#ifndef blockDimy
#define blockDimy 16		
#endif

#ifndef blockDimz
#define blockDimz 1		
#endif
////////////////////////////////////////////////////////////////////////////////////////////////////
/// Mirror effect, acts like Neumann Boundary Condition
#define at(x, y, z, dimx, dimy, dimz) (clamp(z, 0, dimz-1)*dimy*dimx		\
									  +clamp(y, 0, dimy-1)*dimx				\
									  +clamp(x, 0, dimx-1))				
////////////////////////////////////////////////////////////////////////////////////////////////////
/// Do not need to use shared memory because computation is small, and reading is dominated
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void __dxt_forward(
	float2* u, 
	float2* dx,
	int dimx, 
	int dimy, 
	int dimz)
{
	//3D global index
	int3 idx = make_int3(
		blockIdx.x*blockDim.x+threadIdx.x,
		blockIdx.y*blockDim.y+threadIdx.y,
		blockIdx.z*blockDim.z+threadIdx.z);
	
	//Check valid indices
	if (idx.x >= dimx || idx.y >= dimy || idx.z >= dimz)
		return;
	
	//
	dx[at(idx.x, idx.y, idx.z, dimx, dimy, dimz)]
	=  0.5f	*	(u[at(idx.x+1, idx.y, idx.z, dimx, dimy, dimz)]
			    -u[at(idx.x-1, idx.y, idx.z, dimx, dimy, dimz)]);	
}
////////////////////////////////////////////////////////////////////////////////////////////////////
void dxt_forward(
	float2* u, 
	float2* dx,
	int dimx, 
	int dimy, 
	int dimz)
{
	dim3 numBlocks((dimx/blockDimx + ((dimx%blockDimx)?1:0)),
				   (dimy/blockDimy + ((dimy%blockDimy)?1:0)),
				   (dimz/blockDimz + ((dimz%blockDimz)?1:0)) );
	dim3 numThreads(blockDimx, blockDimy, blockDimz);
	__dxt_forward<<<numBlocks, numThreads>>>(u, dx, dimx, dimy, dimz);
}
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void __dxt_inverse(
	float2* u, 
	float2* dx,
	int dimx, 
	int dimy, 
	int dimz)
{
	//3D global index
	int3 idx = make_int3(
		blockIdx.x*blockDim.x+threadIdx.x,
		blockIdx.y*blockDim.y+threadIdx.y,
		blockIdx.z*blockDim.z+threadIdx.z);
	
	//Check valid indices
	if (idx.x >= dimx || idx.y >= dimy || idx.z >= dimz)
		return;
	
	//
	dx[at(idx.x, idx.y, idx.z, dimx, dimy, dimz)]
	=  0.5f	*	(u[at(idx.x-1, idx.y, idx.z, dimx, dimy, dimz)]
			    -u[at(idx.x+1, idx.y, idx.z, dimx, dimy, dimz)]);	
}
////////////////////////////////////////////////////////////////////////////////////////////////////
void dxt_inverse(
	float2* u, 
	float2* dx,
	int dimx, 
	int dimy, 
	int dimz)
{
	dim3 numBlocks((dimx/blockDimx + ((dimx%blockDimx)?1:0)),
				   (dimy/blockDimy + ((dimy%blockDimy)?1:0)),
				   (dimz/blockDimz + ((dimz%blockDimz)?1:0)) );
	dim3 numThreads(blockDimx, blockDimy, blockDimz);
	__dxt_inverse<<<numBlocks, numThreads>>>(u, dx, dimx, dimy, dimz);
}
////////////////////////////////////////////////////////////////////////////////////////////////////
__global__
void __dxt_laplacian(
	float2* u, 
	float2* dx,
	int dimx, 
	int dimy, 
	int dimz)
{
	//3D global index
	int3 idx = make_int3(
		blockIdx.x*blockDim.x+threadIdx.x,
		blockIdx.y*blockDim.y+threadIdx.y,
		blockIdx.z*blockDim.z+threadIdx.z);
	
	//Check valid indices
	if (idx.x >= dimx || idx.y >= dimy || idx.z >= dimz)
		return;
	
	//
	dx[at(idx.x, idx.y, idx.z, dimx, dimy, dimz)]
	=  2.0f*u[at(idx.x,   idx.y, idx.z, dimx, dimy, dimz)]
		   -u[at(idx.x-1, idx.y, idx.z, dimx, dimy, dimz)]
		   -u[at(idx.x+1, idx.y, idx.z, dimx, dimy, dimz)];	
}
////////////////////////////////////////////////////////////////////////////////////////////////////
void dxt_laplacian(
	float2* u, 
	float2* dx,
	int dimx, 
	int dimy, 
	int dimz)
{
	dim3 numBlocks((dimx/blockDimx + ((dimx%blockDimx)?1:0)),
				   (dimy/blockDimy + ((dimy%blockDimy)?1:0)),
				   (dimz/blockDimz + ((dimz%blockDimz)?1:0)) );
	dim3 numThreads(blockDimx, blockDimy, blockDimz);
	__dxt_laplacian<<<numBlocks, numThreads>>>(u, dx, dimx, dimy, dimz);
}
////////////////////////////////////////////////////////////////////////////////////////////////////
void dxt(
	float2* u, 
	float2* dx,
	int dimx, 
	int dimy, 
	int dimz,
	int flag)
{
	switch(flag)
	{
	case DDT_FORWARD:		
		dxt_forward(u, dx, dimx, dimy, dimz);
		break;
	case DDT_INVERSE:		
		dxt_inverse(u, dx, dimx, dimy, dimz);
		break;
	case DDT_LAPLACIAN:		
		dxt_laplacian(u, dx, dimx, dimy, dimz);
		break;
	default:
		break;
	}
}
////////////////////////////////////////////////////////////////////////////////////////////////////
}