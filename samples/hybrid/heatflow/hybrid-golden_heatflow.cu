#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <hip/hip_vector_types.h>
using namespace std;

// ----------------------------------------------------------------------------
#define checkLastError() {                                          				\
	hipError_t error = hipGetLastError();                               			\
	int id; 																		\
	hipGetDevice(&id);																\
	if(error != hipSuccess) {                                         				\
		printf("Cuda failure error in file '%s' in line %i: '%s' at device %d \n",	\
			__FILE__,__LINE__, hipGetErrorString(error), id);			      	 	\
		exit(EXIT_FAILURE);  														\
	}                                                               				\
}
// ----------------------------------------------------------------------------
///////////////////////////////////////////////////////////////////////////////
// Neumann Boundary Condition
#define at(x, y, z, dimx, dimy, dimz) (clamp(z, 0, dimz-1)*dimy*dimx		\
									  +clamp(y, 0, dimy-1)*dimx				\
									  +clamp(x, 0, dimx-1))	 
// ---------------------------------------------------------------------------- 
__global__
void __heatflow(float *src, float *dst, int dimx, int dimy, int dimz)
{
	int  index_1d;
	int3 index_3d;
	index_3d.x	=	blockIdx.x * blockDim.x + threadIdx.x;
	index_3d.y	=	blockIdx.y * blockDim.y + threadIdx.y;
	index_3d.z	=	blockIdx.z * blockDim.z + threadIdx.z;
	
	index_1d 	= index_3d.z * dimy * dimx + 
				  index_3d.y * dimx + 
				  index_3d.x;
	
	
	
	// Store back
	if (index_3d.z < dimz && 
		index_3d.y < dimy && 
		index_3d.x < dimx)
		// dst[index_1d] = src[index_1d];
		dst[at(index_3d.x, index_3d.y, index_3d.z, dimx, dimy, dimz)] 
		= (src[at(index_3d.x+1, index_3d.y+0, index_3d.z+0, dimx, dimy, dimz)] +
		   src[at(index_3d.x-1, index_3d.y+0, index_3d.z+0, dimx, dimy, dimz)] +
		   
		   src[at(index_3d.x+0, index_3d.y+1, index_3d.z+0, dimx, dimy, dimz)] +
		   src[at(index_3d.x+0, index_3d.y-1, index_3d.z+0, dimx, dimy, dimz)] +
		   
		   src[at(index_3d.x+0, index_3d.y+0, index_3d.z+1, dimx, dimy, dimz)] +
		   src[at(index_3d.x+0, index_3d.y+0, index_3d.z-1, dimx, dimy, dimz)]) /6.0f;
}
void heatflow(float *src, float *dst, int dimx, int dimy, int dimz)
{
	dim3 numBlocks((dimx/8 + ((dimx%8)?1:0)),
				   (dimy/8 + ((dimy%8)?1:0)),
				   (dimz/8 + ((dimz%8)?1:0)));
	dim3 numThreads(8, 8, 8);
	__heatflow<<<numBlocks, numThreads>>>(src, dst, dimx, dimy, dimz);
}

// ----------------------------------------------------------------------------
int main(int argc, char** argv)
{
	srand(time(NULL)); // for random number generator
	// Specify dimensions
	const int dimx  = 100;
	const int dimy  = 100;
	const int dimz  = 100;

	const int total = dimx*dimy*dimz;
	
	// Allocate host memory
	float *h_src = new float[total];
	float *h_dst = new float[total];
	
	// Allocate device memory
	float *d_src;
	float *d_dst;
	
	hipMalloc((void**)&d_src, total*sizeof(float));		checkLastError();
	hipMalloc((void**)&d_dst, total*sizeof(float));		checkLastError();
	
	// Initialize the image source
	for(int z=0; z<dimz; z++)
	{
		for(int y=0; y<dimy; y++)
		{
			for(int x=0; x<dimx; x++)
			{
				// h_src[z*dimy*dimx+y*dimx+x] = (float)rand();
				h_src[z*dimy*dimx+y*dimx+x] = (float)(z*dimy*dimx+y*dimx+x);
			}
		}
	}
	// Transferring to the device memory
	hipMemcpy(d_src, h_src, total*sizeof(float), hipMemcpyHostToDevice); checkLastError();
	
	heatflow(d_src, d_dst, dimx, dimy, dimz);

	hipMemcpy(h_dst, d_dst, total*sizeof(float), hipMemcpyDeviceToHost); checkLastError();
	
	// Verify the result
	for(int z=0; z<dimz; z++)
	{
		for(int y=0; y<dimy; y++)
		{
			for(int x=0; x<dimx; x++)
			{
				if(h_src[z*dimy*dimx+y*dimx+x] != h_dst[z*dimy*dimx+y*dimx+x])
				{
					printf("Solution doesnot match at x: %d, y: %d, z: %d\n", x, y, z);
					goto cleanup;
				}
				// else
					// printf("Solution match at x: %d, y: %d, z: %d\n", x, y, z);
			}
		}
	}
	printf("Solution is correct.\n");
cleanup:
	hipFree(d_src);
	hipFree(d_dst);
	free(h_src);
	free(h_dst);
	return 0;
}
