#include "hip/hip_runtime.h"
#include "stencil_3d.hpp"
#include "hip/hip_vector_types.h" 

void stencil_3d(float* deviceSrc, float* deviceDst, int dimx, int dimy, int dimz, int halo, hipStream_t stream);

__global__ 
void __stencil_3d(float* deviceSrc, float* deviceDst, int dimx, int dimy, int dimz, int halo);

void stencil_3d(float* deviceSrc, float* deviceDst, int dimx, int dimy, int dimz, int halo, hipStream_t stream)
{
    dim3 blockDim(16, 8, 1);
    dim3 gridDim(
        (dimx/blockDim.x+((dimx%blockDim.x)?1:0)),
        (dimy/blockDim.y+((dimy%blockDim.y)?1:0)),
        // (dimz/blockDim.z+((dimz%blockDim.z)?1:0)) );
		1); /// Sweep the z dimension, 3D
    size_t sharedMemSize  = (blockDim.x+2*halo)*(blockDim.y+2*halo)*(blockDim.z+2*halo)*sizeof(float);
    __stencil_3d<<<gridDim, blockDim, sharedMemSize, stream>>>
     (deviceSrc, deviceDst, dimx, dimy, dimz, halo);
}

#define at(x, y, z, dimx, dimy, dimz) ( clamp((int)(z), 0, dimz-1)*dimy*dimx+      \
                                        clamp((int)(y), 0, dimy-1)*dimx+           \
                                        clamp((int)(x), 0, dimx-1) )                   
__global__ 
void __stencil_3d(float* deviceSrc, float* deviceDst, int dimx, int dimy, int dimz, int halo)
{
    extern __shared__ float sharedMemSrc[];                     										
    int  shared_index_1d, global_index_1d, index_1d;                                      										
    int3 shared_index_3d, global_index_3d, index_3d;                                      										
	
	float center;
	
	index_3d       =  make_int3(blockIdx.x * blockDim.x+threadIdx.x,                    
								blockIdx.y * blockDim.y+threadIdx.y,                    
								blockIdx.z * blockDim.z+threadIdx.z);                   
	index_1d       =  index_3d.z * dimy * dimx+                                         
					  index_3d.y * dimx+                                                
					  index_3d.x;           
	if((index_3d.x >= dimx)||(index_3d.y>=dimy)||(index_3d.z>dimz))
		return;
    // Multi batch reading here                                                           										
    int3 sharedMemDim    = make_int3(blockDim.x+2*halo,                                   										
                                     blockDim.y+2*halo,                                  										
                                     blockDim.z+2*halo);                                  										
    int  sharedMemSize   = sharedMemDim.x*sharedMemDim.y*sharedMemDim.z;                  										
    int3 blockSizeDim    = make_int3(blockDim.x+0*halo,                                   										
                                     blockDim.y+0*halo,                                   										
                                     blockDim.z+0*halo);                                  										
    int  blockSize        = blockSizeDim.x*blockSizeDim.y*blockSizeDim.z;                  									
    int  numBatches       = sharedMemSize/blockSize+((sharedMemSize%blockSize)?1:0);    



	float result;
	int batch, pass, h;
	//First pass will load entire 3 planes, process and write
	{
		for(batch=0; batch<numBatches; batch++)                                           										
		{                                                                                     										
			shared_index_1d  =  threadIdx.z * blockDim.y * blockDim.x+                      										
								threadIdx.y * blockDim.x+                                   										
								threadIdx.x+                                                										
								blockSize*batch; //Magic is here quantm@unist.ac.kr           										
			shared_index_3d  =  make_int3((shared_index_1d % ((blockDim.y+2*halo)*(blockDim.x+2*halo))) % (blockDim.x+2*halo),		
										  (shared_index_1d % ((blockDim.y+2*halo)*(blockDim.x+2*halo))) / (blockDim.x+2*halo),		
										  (shared_index_1d / ((blockDim.y+2*halo)*(blockDim.x+2*halo))) );      					
			global_index_3d  =  make_int3(blockIdx.x * blockDim.x+shared_index_3d.x-halo, 										
										  blockIdx.y * blockDim.y+shared_index_3d.y-halo, 										
										  blockIdx.z * blockDim.z+shared_index_3d.z-halo);										
			global_index_1d  =  global_index_3d.z * dimy * dimx+                                   								
								global_index_3d.y * dimx+                                   										
								global_index_3d.x;                                            										
			if (shared_index_3d.z < (blockDim.z+2*halo))                                    										
			{                                                                                 										
				if(global_index_3d.z >= 0 && global_index_3d.z < dimz &&                      										
				   global_index_3d.y >= 0 && global_index_3d.y < dimy &&                        									
				   global_index_3d.x >= 0 && global_index_3d.x < dimx)                        										
				{                                                                             										
					sharedMemSrc[at(shared_index_3d.x, shared_index_3d.y, shared_index_3d.z, 
								    sharedMemDim.x, sharedMemDim.y, sharedMemDim.z)] 
					= deviceSrc[global_index_1d];                         
				}                                                                             
			}                                                                                 
		}                                                                                     
		__syncthreads();                                                                  
																							  
		// Stencil  processing here                                                           
		result  = sharedMemSrc[at(threadIdx.x+halo+0, threadIdx.y+halo+0, threadIdx.z+halo+0, sharedMemDim.x, sharedMemDim.y, sharedMemDim.z)];	

																							   
		// Single pass writing here                                                           
		index_3d       =  make_int3(blockIdx.x * blockDim.x+threadIdx.x,                    
									blockIdx.y * blockDim.y+threadIdx.y,                    
									blockIdx.z * blockDim.z+threadIdx.z);                   
		index_1d       =  index_3d.z * dimy * dimx+                                         
						  index_3d.y * dimx+                                                
						  index_3d.x;                                                         
																							   
                                                                                   
		deviceDst[index_1d] = result;                                        
	}
	
	//Second pass, swap the shared memory: middle -> top, bottom -> middle, load the bottom plane, this one iterate from 1 to dimz-1
	for(pass=halo; pass<dimz; pass++)
	{
		center = sharedMemSrc[at(threadIdx.x+halo, threadIdx.y+halo, threadIdx.z+halo+halo,    sharedMemDim.x, sharedMemDim.y, sharedMemDim.z)];
		// break;
		for(batch=0; batch<numBatches; batch++)                                           										
		{                                                                                     										
			shared_index_1d  =  threadIdx.z	* blockDim.y * blockDim.x +                      										
								threadIdx.y * blockDim.x +                                   										
								threadIdx.x +                                                										
								blockSize*batch; //Magic is here quantm@unist.ac.kr           										
			shared_index_3d  =  make_int3((shared_index_1d % ((blockDim.y+2*halo)*(blockDim.x+2*halo))) % (blockDim.x+2*halo),		
										  (shared_index_1d % ((blockDim.y+2*halo)*(blockDim.x+2*halo))) / (blockDim.x+2*halo),		
										  (shared_index_1d / ((blockDim.y+2*halo)*(blockDim.x+2*halo))) );      					
			global_index_3d  =  make_int3(blockIdx.x * blockDim.x+shared_index_3d.x-halo, 										
										  blockIdx.y * blockDim.y+shared_index_3d.y-halo, 										
										  pass       * blockDim.z+shared_index_3d.z-halo);										
			global_index_1d  =  global_index_3d.z * dimy * dimx +                                   								
								global_index_3d.y * dimx +                                   										
								global_index_3d.x;  

								
			///!!! Read next plane
			if ((shared_index_3d.z < (blockDim.z+2*halo)) &&  (shared_index_3d.z > halo))
			// if (shared_index_3d.z < (blockDim.z+2*halo)) 
			{                                                                                 										
				if(global_index_3d.z >= 0 && global_index_3d.z < dimz &&                      										
				   global_index_3d.y >= 0 && global_index_3d.y < dimy &&                        									
				   global_index_3d.x >= 0 && global_index_3d.x < dimx)                        										
				{                                                                             										
					sharedMemSrc[at(shared_index_3d.x, shared_index_3d.y, shared_index_3d.z, 
								    sharedMemDim.x, sharedMemDim.y, sharedMemDim.z)] 
					= deviceSrc[global_index_1d];
				}                                                    						                                                 
			}                                                                                 
		}                                                                                     
		__syncthreads();                                                                  
																							  
		// Stencil  processing here                                                           
		// result  = sharedMemSrc[at(threadIdx.x+halo+0, threadIdx.y+halo+0, threadIdx.z+halo+0, sharedMemDim.x, sharedMemDim.y, sharedMemDim.z)];	
		result  = center;	

																							   
		// Single pass writing here                                                           
		index_3d       =  make_int3(blockIdx.x * blockDim.x+threadIdx.x,                    
									blockIdx.y * blockDim.y+threadIdx.y,                    
									pass       * blockDim.z+threadIdx.z);                   
		index_1d       =  index_3d.z * dimy * dimx+                                         
						  index_3d.y * dimx+                                                
						  index_3d.x;                                                         
																							   
                                                                                     
		deviceDst[index_1d] = result;        

		
	}
	///End kernel
}                                                                                         
