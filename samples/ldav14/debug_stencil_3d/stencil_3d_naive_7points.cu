#include "hip/hip_runtime.h"
#include "stencil_3d.hpp"
#include "hip/hip_vector_types.h" 
#include <stdio.h>

#define DIMX 512
#define DIMY 512
#define DIMZ 512

void stencil_3d_naive_7points(hipPitchedPtr deviceSrc, hipPitchedPtr deviceDst, int dimx, int dimy, int dimz, int halo, hipStream_t stream);

__global__ 
void __stencil_3d_naive_7points(hipPitchedPtr deviceSrc, hipPitchedPtr deviceDst, int dimx, int dimy, int dimz, int halo);

void stencil_3d_naive_7points(hipPitchedPtr deviceSrc, hipPitchedPtr deviceDst, int dimx, int dimy, int dimz, int halo, hipStream_t stream)
{
    // dim3 blockDim(32, 8, 1);
	// dim3 blockSize(32, 8, 1);
	dim3 blockDim(8, 8, 8);
	dim3 blockSize(8, 8, 8);
    dim3 gridDim(
        (dimx/blockSize.x+((dimx%blockSize.x)?1:0)),
        (dimy/blockSize.y+((dimy%blockSize.y)?1:0)),
        (dimz/blockSize.z+((dimz%blockSize.z)?1:0)) );
        // 1);
	size_t sharedMemSize  = (blockSize.x+2*halo)*(blockSize.y+2*halo)*(blockSize.z+2*halo)*sizeof(float);
    __stencil_3d_naive_7points<<<gridDim, blockDim, sharedMemSize, stream>>>
		(deviceSrc, deviceDst, dimx, dimy, dimz, halo);
}

#define at(x, y, z, dimx, dimy, dimz) ( clamp((z), 0, dimz-1)*dimy*dimx+      \
                                        clamp((y), 0, dimy-1)*dimx+           \
                                        clamp((x), 0, dimx-1) )                

__global__ 
void __stencil_3d_naive_7points(hipPitchedPtr deviceSrc, hipPitchedPtr deviceDst, int dimx, int dimy, int dimz, int halo)
{
	int3 index_3d = make_int3(blockDim.x * blockIdx.x + threadIdx.x,
							  blockDim.y * blockIdx.y + threadIdx.y,
							  blockDim.z * blockIdx.z + threadIdx.z);
	char* d_src = (char*)deviceSrc.ptr; 
	char* d_dst = (char*)deviceDst.ptr; 
	size_t pitch = deviceSrc.pitch; 
	size_t slicePitch = pitch * dimy;
	
	char* sliceSrc;
	char* sliceDst;
	float* rowSrc;
	float* rowDst;
	
	float result, tmp, alpha, beta;
	beta = 0.0625f;
	alpha = 0.1f;	
	// float result;
	
	// if(((index_3d.z >0) && (index_3d.z < (dimz-1))) &&
	   // ((index_3d.y >0) && (index_3d.y < (dimy-1))) &&
	   // ((index_3d.x >0) && (index_3d.x < (dimx-1))) )
	// {
		
		// for(int zz=-1; zz<2; zz++)
		// {
			// sliceSrc = d_src + (index_3d.z + zz) * slicePitch; 
			// for(int yy=-1; yy<2; yy++)
			// {
				// rowSrc = (float*)(sliceSrc + (index_3d.y + yy) * pitch); 
				// for(int xx=-1; xx<2; xx++)
				// {
					// if((zz!=0) && (yy!=0) && (xx!=0))
					// {
						// tmp += rowSrc[index_3d.x + xx];
					// }
				// }
			// }
		// }
	// }
	
	// Debug: Write back to the device Result 1 to 1
	// sliceSrc = d_src + index_3d.z * slicePitch; 	
	// rowSrc = (float*)(sliceSrc + index_3d.y * pitch); 
	// result = rowSrc[index_3d.x];
	
	
	// sliceDst = d_dst + index_3d.z * slicePitch; 	
	// rowDst = (float*)(sliceDst + index_3d.y * pitch); 
	// rowDst[index_3d.x] = result;
	// return;
	
	// Debug: Stencil no shared mem

	
	// Debug: Stencil with shared mem
	extern __shared__ float sharedMem[];                     										
	// __shared__ float sharedMem[34][10][3];                     										
	int3 opened_index_3d, closed_index_3d, offset_index_3d, global_index_3d;
	int  opened_index_1d, closed_index_1d, offset_index_1d, global_index_1d;
	int3 openedDim,  closedDim;
	int  openedSize, closedSize;
	int  thisReading, thisWriting;
	int  numThreads, numReading, numWriting, batch, sweep;
	
	// for(sweep=0; sweep<dimz; sweep++)
	// {
	
	//Calculate the closed form, instruction parallelism
	closedDim  = make_int3(1*blockDim.x,
	 		 		       1*blockDim.y,
						   1*blockDim.z);
	openedDim  = make_int3(closedDim.x + 2*halo,
	 					   closedDim.y + 2*halo,
						   closedDim.z + 2*halo);
						  
	offset_index_3d  = make_int3(blockIdx.x * closedDim.x, 
								 blockIdx.y * closedDim.y,
								 blockIdx.z * closedDim.z);
								 // sweep * closedDim.z);
	///
	numThreads = blockDim.x  * blockDim.y  * blockDim.z;
	openedSize = openedDim.x * openedDim.y * openedDim.z;
	closedSize = closedDim.x * closedDim.y * closedDim.z;
	
	///
	numReading = (openedSize / numThreads) + ((openedSize % numThreads)?1:0);    
	numWriting = (closedSize / numThreads) + ((closedSize % numThreads)?1:0);    
	
	
	#pragma unroll
	for(thisReading=0; thisReading<numReading; thisReading++)
	{
		opened_index_1d =  threadIdx.z * blockDim.y * blockDim.x +                      										
						   threadIdx.y * blockDim.x +                                   										
						   threadIdx.x +                  
						   thisReading * numThreads; //Flatten everything
		opened_index_3d = make_int3((opened_index_1d % (openedDim.y*openedDim.x) % openedDim.x),		
								    (opened_index_1d % (openedDim.y*openedDim.x) / openedDim.x),		
									(opened_index_1d / (openedDim.y*openedDim.x)) );  
		global_index_3d = make_int3((offset_index_3d.x + opened_index_3d.x - 1*halo),
									(offset_index_3d.y + opened_index_3d.y - 1*halo),
									(offset_index_3d.z + opened_index_3d.z - 1*halo) );
		global_index_1d = global_index_3d.z * dimy * dimx +
						  global_index_3d.y * dimx +
						  global_index_3d.x;
		// if(global_index_1d == 0) printf("numReading (%d), numWriting (%d) \n", numReading, numWriting);
		if (opened_index_3d.z < openedDim.z)
		{
			if(global_index_3d.z >= 0 && global_index_3d.z < dimz &&	
			   global_index_3d.y >= 0 && global_index_3d.y < dimy &&
			   global_index_3d.x >= 0 && global_index_3d.x < dimx) 
			{
				sliceSrc = d_src + (global_index_3d.z) * slicePitch; 
				rowSrc = (float*)(sliceSrc + (global_index_3d.y) * pitch); 
				sharedMem[at(opened_index_3d.x, 
							 opened_index_3d.y, 
							 opened_index_3d.z,
						     openedDim.x, 
							 openedDim.y, 
							 openedDim.z)]
				= rowSrc[global_index_3d.x];
			}
		}
		__syncthreads();	
	}
	
	
	#pragma unroll
	for(thisWriting=0; thisWriting<numWriting; thisWriting++)
	{
		closed_index_1d =  threadIdx.z * blockDim.y * blockDim.x +                      										
						   threadIdx.y * blockDim.x +                                   										
						   threadIdx.x +                  
						   thisWriting * numThreads; //Magic is here 
		closed_index_3d = make_int3((closed_index_1d % (closedDim.y*closedDim.x) % closedDim.x),		
								    (closed_index_1d % (closedDim.y*closedDim.x) / closedDim.x),		
									(closed_index_1d / (closedDim.y*closedDim.x)) );  
		global_index_3d = make_int3((offset_index_3d.x + closed_index_3d.x),
									(offset_index_3d.y + closed_index_3d.y),
									(offset_index_3d.z + closed_index_3d.z) );
		global_index_1d = global_index_3d.z * dimy * dimx +
						  global_index_3d.y * dimx +
						  global_index_3d.x;
						  
						  
		result	= sharedMem[at(closed_index_3d.x + 1*halo + 0, 
							   closed_index_3d.y + 1*halo + 0, 
							   closed_index_3d.z + 1*halo + 0,
						       openedDim.x, 
							   openedDim.y, 
							   openedDim.z)];
								
		if (closed_index_3d.z < closedDim.z)
		{
			if(global_index_3d.z >= 0 && global_index_3d.z < dimz &&	
			   global_index_3d.y >= 0 && global_index_3d.y < dimy &&
			   global_index_3d.x >= 0 && global_index_3d.x < dimx) 
			{
				// deviceDst[global_index_1d] = result;
				sliceDst = d_dst + closed_index_3d.z * slicePitch; 	
				rowDst = (float*)(sliceDst + closed_index_3d.y * pitch); 
				rowDst[closed_index_3d.x] = result;
			}
		}
	}
	
	

	// sliceSrc = d_src + (index_3d.z) * slicePitch; 
	// rowSrc = (float*)(sliceSrc + (index_3d.y) * pitch); 
	// result = alpha*rowSrc[index_3d.x] + beta*tmp;	
	
	
	
	// } //End sweep
	
	// Write back to the device Result
	// sliceDst = d_dst + index_3d.z * slicePitch; 	
	// rowDst = (float*)(sliceDst + index_3d.y * pitch); 
	// rowDst[index_3d.x] = result;
	
	
	// /// Naive copy
	// // d_dst[index_3d.z][index_3d.y][index_3d.x] 
	// // = d_src[index_3d.z][index_3d.y][index_3d.x];
	// char* d_src = (char*)deviceSrc.ptr; 
	// char* d_dst = (char*)deviceDst.ptr; 
	// size_t pitch = deviceSrc.pitch; 
	// size_t slicePitch = pitch * dimy;
	// // for (int z = 0; z < dimz; ++z) 
	// // { 
		// char* sliceSrc = d_src + index_3d.z * slicePitch; 
		// char* sliceDst = d_dst + index_3d.z * slicePitch; 
		// // for (int y = 0; y < dimy; ++y) 
		// // { 
			// float* rowSrc = (float*)(sliceSrc + index_3d.y * pitch); 
			// float* rowDst = (float*)(sliceDst + index_3d.y * pitch); 
			// // for (int x = 0; x < dimx; ++x) 
			// // { 
				// rowDst[index_3d.x] = rowSrc[index_3d.x];
			// // } 
		// // } 
	// // }
}                                                                                         
