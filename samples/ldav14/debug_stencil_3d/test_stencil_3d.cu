#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <sstream>
#include <iomanip>      // std::setfill, std::setw
#include <string>
#include <sys/ioctl.h>
#include <hip/hip_runtime.h>
#include <gpu_timer.hpp>
#include <hetero_cmdparser.hpp>
#include "stencil_3d.hpp"

using namespace std;
////////////////////////////////////////////////////////////////////////////////////////////////////
#define checkLastError() {                                          				\
	hipError_t error = hipGetLastError();                               			\
	int id; 																		\
	hipGetDevice(&id);																\
	if(error != hipSuccess) {                                         				\
		printf("Cuda failure error in file '%s' in line %i: '%s' at device %d \n",	\
			__FILE__,__LINE__, hipGetErrorString(error), id);			      	 	\
		exit(EXIT_FAILURE);  														\
	}                                                               				\
}
////////////////////////////////////////////////////////////////////////////////////////////////////
#define checkWriteFile(filename, pData, size) {                    				\
		fstream *fs = new fstream;												\
		fs->open(filename, ios::out|ios::binary);								\
		if (!fs->is_open())														\
		{																		\
			fprintf(stderr, "Cannot open file '%s' in file '%s' at line %i\n",	\
			filename, __FILE__, __LINE__);										\
			return 1;															\
		}																		\
		fs->write(reinterpret_cast<char*>(pData), size);						\
		fs->close();															\
		delete fs;																\
	}
////////////////////////////////////////////////////////////////////////////////////////////////////


#define at(x, y, z, dimx, dimy, dimz) ( clamp((int)(z), 0, dimz-1)*dimy*dimx +       \
                                        clamp((int)(y), 0, dimy-1)*dimx +            \
                                        clamp((int)(x), 0, dimx-1) )                   
__global__ 
void __copy_3d(float* deviceSrc, float* deviceDst, int dimx, int dimy, int dimz, int halo)
{
	
	// Single pass writing here                                                           
	int3 index_3d       =  make_int3(blockIdx.x * blockDim.x + threadIdx.x,                    
								blockIdx.y * blockDim.y + threadIdx.y,                    
								blockIdx.z * blockDim.z + threadIdx.z);                   
	int index_1d       =  index_3d.z * dimy * dimx +                                          
					  index_3d.y * dimx +                                                 
					  index_3d.x;                                                         
																						   
	if (index_3d.z < dimz &&                                                              
		index_3d.y < dimy &&                                                              
		index_3d.x < dimx)                                                                
	{                                                                                     
		deviceDst[index_1d] = deviceSrc[index_1d];                                        
	} 


}  
void copy_3d(float* deviceSrc, float* deviceDst, int dimx, int dimy, int dimz, int halo, hipStream_t stream)
{
    dim3 blockDim(32, 4, 4);
    dim3 gridDim(
        (dimx/blockDim.x + ((dimx%blockDim.x)?1:0)),
        (dimy/blockDim.y + ((dimy%blockDim.y)?1:0)),
        (dimz/blockDim.z + ((dimz%blockDim.z)?1:0)) );
		// 1); /// Sweep the z dimension, 3D
    // size_t sharedMemSize  = (blockDim.x+2*halo)*(blockDim.y+2*halo)*(blockDim.z+2*halo)*sizeof(float);
    __copy_3d<<<gridDim, blockDim, 0, stream>>>
     (deviceSrc, deviceDst, dimx, dimy, dimz, halo);
}                                                                                       

////////////////////////////////////////////////////////////////////////////////////////////////////
texture<float, 3, hipReadModeElementType> tex; 
////////////////////////////////////////////////////////////////////////////////////////////////////
const char* key =
	"{ h   |help    |      | print help message }"	
	"{ dx  |dimx    | 512  | dimensionx }"
	"{ dy  |dimy    | 512  | dimensiony }"
	"{ dz  |dimz    | 512  | dimensionz }"
	"{ bx  |blockx  | 4    | blockDimx }"
	"{ by  |blocky  | 4    | blockDimy }"
	"{ bz  |blockz  | 1    | blockDimz }"
	"{ ilp |istrlp  | 1    | instruction parallelism factor }"
	"{ num |num     | 20   | numLoops }"
	;
int main(int argc, char **argv)
{
	srand(time(NULL)); // for random number generator
	
	// Parsing the arguments
	CommandLineParser cmd(argc, argv, key);
	const int numTrials			= cmd.get<int>("num", false);
	const int dimx  			= cmd.get<int>("dimx", false);
	const int dimy  			= cmd.get<int>("dimy", false);
	const int dimz  			= cmd.get<int>("dimz", false);

	const int total = dimx*dimy*dimz;
	
	const int bx  			= cmd.get<int>("bx", false);
	const int by  			= cmd.get<int>("by", false);
	const int bz  			= cmd.get<int>("bz", false);
	
	const int ilp  			= cmd.get<int>("ilp", false);
	
	hipSetDevice(0);
	hipDeviceReset();
	// Specify dimensions
	
	// Allocate host memory
	float *h_src = new float[total];
	float *h_dst = new float[total];
	
	// Allocate device memory
	// float *d_src;
	// float *d_dst;
	// hipMalloc((void**)&d_src, total*sizeof(float));		checkLastError();
	// hipMalloc((void**)&d_dst, total*sizeof(float));		checkLastError();
	
	// Initialize the image source
	for(int z=0; z<dimz; z++)
	{
		for(int y=0; y<dimy; y++)
		{
			for(int x=0; x<dimx; x++)
			{
				h_src[z*dimy*dimx+y*dimx+x] = (float)rand();
			}
		}
	}
	
		
	///!!! Setting coefficients here
	// float a = -6.0f;
  	// float b = +0.1f;
	
	///!!! Setting texture parameter here
	// hipExtent volumeSize = make_hipExtent(dimx, dimy, dimz);

	tex.normalized = false;      //Donot normalize to [0, 1]
	
	// hipFilterModePoint 	 	 Point filter mode
	// hipFilterModeLinear 	 Linear filter mode
    tex.filterMode = hipFilterModePoint;      // linear interpolation

	// hipAddressModeWrap 	 	 Wrapping address mode
	// hipAddressModeClamp 	 Clamp to edge address mode
	// hipAddressModeMirror 	 Mirror address mode
	// hipAddressModeBorder 	 Border address mode
	tex.addressMode[0] = hipAddressModeMirror;  
    tex.addressMode[1] = hipAddressModeMirror ;
    tex.addressMode[2] = hipAddressModeMirror ;
	
	
	
	///!!! Allocate and copy to device memory
	hipExtent volumeSize = make_hipExtent(dimx, dimy, dimz);
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();	checkLastError();
	hipArray *d_src;
	hipMalloc3DArray(&d_src, &channelDesc, volumeSize);				checkLastError();
	// hipArray *d_dst;
	// hipMalloc3DArray(&d_src, &channelDesc, volumeSize);				checkLastError();
	float *d_dst;
	hipMalloc((void**)&d_dst, total*sizeof(float));		checkLastError();
	
	
	cout << __FILE__ << " " << __LINE__ << endl;
	
	///!!! Transferring to the device memory
	// hipMemcpy(d_src, h_src, total*sizeof(float), hipMemcpyHostToDevice); checkLastError();
	hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr   = make_hipPitchedPtr(h_src, volumeSize.width*sizeof(float), volumeSize.width, volumeSize.height);
    copyParams.dstArray = d_src;
    copyParams.extent   = volumeSize;
    copyParams.kind     = hipMemcpyHostToDevice;
    hipMemcpy3D(&copyParams);	checkLastError();


	///!!! Bind the texture memory
	hipBindTextureToArray(tex, d_src, channelDesc); checkLastError();
	
	

	
	
	GpuTimer gpu_timer;
	gpu_timer.Start();
	for(int n=0; n<numTrials; n++)
		stencil_3d(d_src, d_dst, dimx, dimy, dimz, 1);
	gpu_timer.Stop();
	
	///!!! Normalize the running time
	float ms = gpu_timer.Elapsed()/numTrials; 
	printf("Time %4.3f ms\n", ms);	

	int numOperations, GFLOPS;
	numOperations = 8;
	GFLOPS 		  = (float)total*(float)numOperations* 1.0e-9f/(ms*1.0e-3f);
	printf("Performance of %s is %04.4f   GFLOPS/s\n", argv[0],  GFLOPS); 
	
	
	///!!! Check correctness here
	// hipMemcpy(h_dst, d_dst, total*sizeof(float), hipMemcpyDeviceToHost); checkLastError();
	
	///!!! Pring a line to terminate
	struct winsize w;
    ioctl(0, TIOCGWINSZ, &w);
	for(int k=0; k<w.ws_col; k++) 
		printf("-");
	printf("\n");
	checkLastError();

	
	
	// // Verify the result
	// for(int z=0; z<dimz; z++)
	// {
		// for(int y=0; y<dimy; y++)
		// {
			// for(int x=0; x<dimx; x++)
			// {
				// if(h_src[z*dimy*dimx+y*dimx+x] != h_dst[z*dimy*dimx+y*dimx+x])
				// {
					// printf("Solution doesnot match at x: %d, y: %d, z: %d\n", x, y, z);
					// goto cleanup;
				// }
			// }
		// }
	// }
	// printf("Solution is correct.\n");
// cleanup:
	hipFree(d_src);
	hipFree(d_dst);
	free(h_src);
	free(h_dst);
	return 0;
}